// Familiarize with memory allocs


#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "time.h"

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = b[blockIdx.x] + a[blockIdx.x];
}

void random_int(int *arr, int size) {
    srand(time(NULL));
    for (int i = 0; i < size; i++) {
        arr[i] = rand();
    }

}

#define N 1024
int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a, d_b, d_c); 

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    free(a); free(b); free(c);
    hipFree(a); hipFree(b); hipFree(c);

    hipDeviceSynchronize();
    return 0;
}