#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void cuda_hello(){
    println("Hello World!");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}